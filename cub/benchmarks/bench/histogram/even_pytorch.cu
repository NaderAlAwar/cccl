/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "histogram_common.cuh"
#include "npy.hpp"
#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS ipt 4:28:1
// %RANGE% TUNE_THREADS tpb 128:1024:32
// %RANGE% TUNE_RLE_COMPRESS rle 0:1:1
// %RANGE% TUNE_WORK_STEALING ws 0:1:1
// %RANGE% TUNE_MEM_PREFERENCE mem 0:2:1
// %RANGE% TUNE_LOAD ld 0:2:1
// %RANGE% TUNE_LOAD_ALGORITHM_ID laid 0:2:1
// %RANGE% TUNE_VEC_SIZE_POW vec 0:2:1

hipError_t check(hipError_t status)
{
  if (hipSuccess == status)
  {
    return hipSuccess;
  }

  std::stringstream ss;

  ss << "Cuda error '";
  ss << hipGetErrorString(status);
  ss << "' encountered on line ";

  throw std::runtime_error(ss.str());
}

template <typename SampleT, typename CounterT, typename OffsetT>
static void even(nvbench::state& state, nvbench::type_list<SampleT, CounterT, OffsetT>)
{
  constexpr int num_channels        = 1;
  constexpr int num_active_channels = 1;

  using sample_iterator_t = SampleT*;
  using LevelT            = int;

#if !TUNE_BASE
  using policy_t = policy_hub_t<key_t, num_channels, num_active_channels>;
  using dispatch_t =
    hipcub::DispatchHistogram<num_channels, //
                           num_active_channels,
                           sample_iterator_t,
                           CounterT,
                           LevelT,
                           OffsetT,
                           policy_t>;
#else // TUNE_BASE
  using dispatch_t =
    hipcub::DispatchHistogram<num_channels, //
                           num_active_channels,
                           sample_iterator_t,
                           CounterT,
                           /* LevelT = */ LevelT,
                           OffsetT>;
#endif // TUNE_BASE

  // const auto entropy   = str_to_entropy(state.get_string("Entropy"));
  const auto elements  = state.get_int64("Elements{io}");
  const auto num_bins  = 256;
  const int num_levels = static_cast<int>(num_bins) + 1;

  // const SampleT lower_level = 0;
  // const SampleT upper_level = get_upper_level<SampleT, OffsetT>(num_bins, elements);
  const LevelT lower_level = 0;
  const LevelT upper_level = 256;

  // thrust::device_vector<SampleT> input = generate(elements, entropy, lower_level, upper_level);

  std::string file_name = "data.npy";
  auto npy_data         = npy::read_npy<uint8_t>(file_name);

  size_t n_elems = npy_data.data.size();
  std::cout << "Read " << n_elems << " elements from " << file_name << std::endl;

  thrust::device_vector<uint8_t> input(n_elems);

  hipError_t err = check(hipMemcpy(
    static_cast<void*>(thrust::raw_pointer_cast(input.data())),
    static_cast<void*>(thrust::raw_pointer_cast(npy_data.data.data())),
    npy_data.data.size() * sizeof(uint8_t),
    hipMemcpyHostToDevice));

  thrust::device_vector<CounterT> hist(num_bins);

  SampleT* d_input      = thrust::raw_pointer_cast(input.data());
  CounterT* d_histogram = thrust::raw_pointer_cast(hist.data());

  std::uint8_t* d_temp_storage = nullptr;
  std::size_t temp_storage_bytes{};

  cuda::std::bool_constant<sizeof(SampleT) == 1> is_byte_sample;
  OffsetT num_row_pixels     = static_cast<OffsetT>(elements);
  OffsetT num_rows           = 1;
  OffsetT row_stride_samples = num_row_pixels;

  state.add_element_count(elements);
  state.add_global_memory_reads<SampleT>(elements);
  state.add_global_memory_writes<CounterT>(num_bins);

  dispatch_t::DispatchEven(
    d_temp_storage,
    temp_storage_bytes,
    d_input,
    {d_histogram},
    {num_levels},
    {lower_level},
    {upper_level},
    num_row_pixels,
    num_rows,
    row_stride_samples,
    0,
    is_byte_sample);

  thrust::device_vector<nvbench::uint8_t> tmp(temp_storage_bytes);
  d_temp_storage = thrust::raw_pointer_cast(tmp.data());

  state.exec(nvbench::exec_tag::gpu | nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::DispatchEven(
      d_temp_storage,
      temp_storage_bytes,
      d_input,
      {d_histogram},
      {num_levels},
      {lower_level},
      {upper_level},
      num_row_pixels,
      num_rows,
      row_stride_samples,
      launch.get_stream(),
      is_byte_sample);
  });
}

using counter_types     = nvbench::type_list<int32_t>;
using some_offset_types = nvbench::type_list<int32_t>;

using sample_types = nvbench::type_list<uint8_t>;

NVBENCH_BENCH_TYPES(even, NVBENCH_TYPE_AXES(sample_types, counter_types, some_offset_types))
  .set_name("base")
  .set_type_axes_names({"SampleT{ct}", "CounterT{ct}", "OffsetT{ct}"})
  .add_int64_axis("Elements{io}", {10485760});
